
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#define filterWidth 3
#define filterHeight 3

void checkCUDAError(hipError_t cudaStatus, const char* errorMessage) {
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "%s: %s\n", errorMessage, hipGetErrorString(cudaStatus));
        exit(1);
    }
}

__global__ void filter(const unsigned int* input_image, unsigned int* output_image, unsigned int height, unsigned int width) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int filter[filterWidth][filterHeight] = {
        {1, 1, 1},
        {1, 3, 1},
        {1, 1, 1}
    };

    int filter_sum = 0;
    for (int i = 0; i < filterWidth; i++) {
        for (int j = 0; j < filterHeight; j++) {
            filter_sum += filter[i][j];
        }
    }

    int sum = 0;

    if (y == 0 && x == 0) { // Top-left corner
        sum += input_image[y * width + x] * filter[1][1];
        sum += input_image[y * width + (x + 1)] * filter[1][2];
        sum += input_image[(y + 1) * width + x] * filter[2][1];
        sum += input_image[(y + 1) * width + (x + 1)] * filter[2][2];
    } else if (y == 0 && x == width - 1) { // Top-right corner
        sum += input_image[y * width + x] * filter[1][1];
        sum += input_image[y * width + (x - 1)] * filter[1][0];
        sum += input_image[(y + 1) * width + x] * filter[2][1];
        sum += input_image[(y + 1) * width + (x - 1)] * filter[2][0];
    } else if (y == height - 1 && x == 0) { // Bottom-left corner
        sum += input_image[y * width + x] * filter[1][1];
        sum += input_image[y * width + (x + 1)] * filter[1][2];
        sum += input_image[(y - 1) * width + x] * filter[0][1];
        sum += input_image[(y - 1) * width + (x + 1)] * filter[0][2];
    } else if (y == height - 1 && x == width - 1) { // Bottom-right corner
        sum += input_image[y * width + x] * filter[1][1];
        sum += input_image[y * width + (x - 1)] * filter[1][0];
        sum += input_image[(y - 1) * width + x] * filter[0][1];
        sum += input_image[(y - 1) * width + (x - 1)] * filter[0][0];
    } else if (y == 0) { // Top edge
        sum += input_image[y * width + x] * filter[1][1];
        sum += input_image[y * width + (x - 1)] * filter[1][0];
        sum += input_image[y * width + (x + 1)] * filter[1][2];
        sum += input_image[(y + 1) * width + x] * filter[2][1];
        sum += input_image[(y + 1) * width + (x - 1)] * filter[2][0];
        sum += input_image[(y + 1) * width + (x + 1)] * filter[2][2];
    } else if (y == height - 1) { // Bottom edge
        sum += input_image[y * width + x] * filter[1][1];
        sum += input_image[y * width + (x - 1)] * filter[1][0];
        sum += input_image[y * width + (x + 1)] * filter[1][2];
        sum += input_image[(y - 1) * width + x] * filter[0][1];
        sum += input_image[(y - 1) * width + (x - 1)] * filter[0][0];
        sum += input_image[(y - 1) * width + (x + 1)] * filter[0][2];
    } else if (x == 0) { // Left edge
        sum += input_image[y * width + x] * filter[1][1];
        sum += input_image[(y - 1) * width + x] * filter[0][1];
        sum += input_image[(y + 1) * width + x] * filter[2][1];
        sum += input_image[y * width + (x + 1)] * filter[1][2];
        sum += input_image[(y - 1) * width + (x + 1)] * filter[0][2];
        sum += input_image[(y + 1) * width + (x + 1)] * filter[2][2];
    } else if (x == width - 1) { // Right edge
        sum += input_image[y * width + x] * filter[1][1];
        sum += input_image[(y - 1) * width + x] * filter[0][1];
        sum += input_image[(y + 1) * width + x] * filter[2][1];
        sum += input_image[y * width + (x - 1)] * filter[1][0];
        sum += input_image[(y - 1) * width + (x - 1)] * filter[0][0];
        sum += input_image[(y + 1) * width + (x - 1)] * filter[2][0];
    } else { // Center
        sum += input_image[(y - 1) * width + (x - 1)] * filter[0][0];
        sum += input_image[(y - 1) * width + x] * filter[0][1];
        sum += input_image[(y - 1) * width + (x + 1)] * filter[0][2];
        sum += input_image[y * width + (x - 1)] * filter[1][0];
        sum += input_image[y * width + x] * filter[1][1];
        sum += input_image[y * width + (x + 1)] * filter[1][2];
        sum += input_image[(y + 1) * width + (x - 1)] * filter[2][0];
        sum += input_image[(y + 1) * width + x] * filter[2][1];
        sum += input_image[(y + 1) * width + (x + 1)] * filter[2][2];
    }

    output_image[y * width + x] = sum / filter_sum;
}


int main() {

    // Image size
    unsigned int height = 1024;
    unsigned int width = 1920;
    size_t size = height * width * sizeof(unsigned int);

    // Allocate memory for images on host
    unsigned int* h_input_image = (unsigned int*)malloc(size);
    unsigned int* h_output_image = (unsigned int*)malloc(size);

    if (h_input_image == NULL || h_output_image == NULL) {
        fprintf(stderr, "Failed to allocate host memory.\n");
        exit(1);
    }

    // Initialize input image from file
    FILE* file = fopen("original_0.csv", "r");
    if (file == NULL) {
        fprintf(stderr, "Failed to open input file.\n");
        exit(1);
    }

    char line[10240];
    unsigned int row = 0;

    while (fgets(line, sizeof(line), file) && row < height) {
        char *token;
        unsigned int col = 0;

        token = strtok(line, ",");
        while (token != NULL && col < width) {
            h_input_image[row * width + col] = atoi(token);  // Use atoi to convert string to int
            token = strtok(NULL, ",");
            col++;
        }
        row++;
    }

    fclose(file);

    // Initialize output image
    for (unsigned int i = 0; i < height; i++) {
        for (unsigned int j = 0; j < width; j++) {
            h_output_image[i * width + j] = 0;
        }
    }

    // Allocate memory for images on device
    unsigned int* d_input_image;
    unsigned int* d_output_image;
    hipError_t cudaStatus;

    cudaStatus = hipMalloc(&d_input_image, size);
    checkCUDAError(cudaStatus, "hipMalloc failed for input image");

    cudaStatus = hipMalloc(&d_output_image, size);
    checkCUDAError(cudaStatus, "hipMalloc failed for output image");

    // Copy input image from host to device
    cudaStatus = hipMemcpy(d_input_image, h_input_image, size, hipMemcpyHostToDevice);
    checkCUDAError(cudaStatus, "hipMemcpy failed from host to device");

    // Define block and grid sizes
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    // Measure execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start recording
    hipEventRecord(start);

    // Launch the filter kernel
    filter<<<gridSize, blockSize>>>(d_input_image, d_output_image, height, width);

    cudaStatus = hipGetLastError();
    checkCUDAError(cudaStatus, "Kernel launch failed");

    // Stop recording
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    float microseconds = milliseconds * 1000.0f; // Convert milliseconds to microseconds
    float nanoseconds = microseconds * 1000.0f; // Convert microseconds to nanoseconds
    //printf("Execution time: %f", nanoseconds);

    printf("Execution time: %f nanoseconds\n", nanoseconds);

    // Copy output image from device to host
    cudaStatus = hipMemcpy(h_output_image, d_output_image, size, hipMemcpyDeviceToHost);
    checkCUDAError(cudaStatus, "hipMemcpy failed from device to host");

    // Save output image to CSV file
    FILE* outFile = fopen("result.csv", "w");
    if (outFile == NULL) {
        fprintf(stderr, "Failed to open result file.\n");
        exit(1);
    }

    for (unsigned int i = 0; i < height; i++) {
        for (unsigned int j = 0; j < width; j++) {
            fprintf(outFile, "%d", h_output_image[i * width + j]);
            if (j < width - 1) {
                fprintf(outFile, ",");
            }
        }
        fprintf(outFile, "\n");
    }

    fclose(outFile);

    // Free device memory
    hipFree(d_input_image);
    hipFree(d_output_image);

    // Free host memory
    free(h_input_image);
    free(h_output_image);

    return 0;
}
